
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
  int pitch_a, int pitch_b, int pitch_c,
  int n, int m, int p )
{
    int i = blockIdx.x*32 + threadIdx.x;
    int j = blockIdx.y;

    float sum = 0.0;
    for( int k = 0; k < p; ++k ) // p iterations
      sum += b[i+pitch_b*k] * c[k+pitch_c*j]; // 2 flops. 2p mem accesses per thread.
    a[i+pitch_a*j] = sum; //1 more global memory access
}
