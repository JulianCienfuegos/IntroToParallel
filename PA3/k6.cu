
#include <hip/hip_runtime.h>
/*
We want to use shared memory, and access coalesced global memory.
*/

extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
        int pitch_a, int pitch_b, int pitch_c,
        int n, int m, int p )
{
        int i = blockIdx.x*64 + threadIdx.x;
        int j = blockIdx.y*2;
        float sum0 = 0.0, sum1 = 0.0, sum2=0.0, sum3=0.0;

        for(int k = 0; k < p; ++k)
        {
                float c_tmp = c[k*pitch_c+i];
                float c_tmp2 = c[k*pitch_c+i+32];
                float b_tmp = b[j+pitch_b*k];
                float b_tmp2 = b[j+1+pitch_b*k];
                sum0 += b_tmp*c_tmp;
                sum1 += b_tmp2*c_tmp;
                sum2 += b_tmp*c_tmp2;
                sum3 += b_tmp2*c_tmp2;
        }

        a[j+pitch_a*i] = sum0;
        a[j+pitch_a*(i+32)] = sum1;
        a[j+1+pitch_a*i]=sum2;
        a[j+1+pitch_a*(i+32)] =sum3;
}


